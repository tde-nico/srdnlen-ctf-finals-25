#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <map>



// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
#define CHECK(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t res, const char *func, const char *file, const int line)
{
	if (!res)
		return ;
	std::cerr << "CUDA error = " << static_cast<unsigned int>(res);
	std::cerr << " at " << file << ":" << line << " '" << func << "' \n";
	hipDeviceReset();
	exit(1);
}

typedef unsigned long long ll;

__device__ unsigned char constant_bits[] = {
	0x01, 0x00, 0x01, 0x01, 0x00, 0x01, 0x00, 0x00, 0x01, 0x00, 
	0x01, 0x01, 0x01, 0x00, 0x00, 0x01, 0x01, 0x00, 0x01, 0x01, 
	0x00, 0x00, 0x01, 0x00, 0x00, 0x01, 0x01, 0x00, 0x01, 0x00, 
	0x01, 0x00, 0x01, 0x00, 0x00, 0x01, 0x01, 0x01, 0x00, 0x01, 
	0x00, 0x00, 0x00, 0x01, 0x01, 0x01, 0x00, 0x01, 0x00, 0x00, 
	0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 
	0x00, 0x00, 0x00, 0x00
};


#define STEP(x) ((((x ^ (x >> 12)) << 25) ^ x ^ (x >> 12)) >> 27) ^ ((x ^ (x >> 12)) << 25) ^ x ^ (x >> 12)


// time:    0000000068d57f89
// pid:     00000000000f8c43
// address: 00007f95168f46c0
// seed:    00007f957e55b50a
// 0dbdb1da008ce6a4e8b46f0f83aa8752b3862504f0b5dc69a0d441baa306e04e3caee5e572b041fab2db55f9b521372cbd4088ad346a97815906164b77b70003f078b0e51f4b44234efcb468a464e561f88182f00ad172650dab317d4279a2f6ecea1135a62a85b1a4293783cb3a28df36a992b0111060f858d50912d0bc504db44f38535511e319ac1249d1b76156a60246462a959f5aed9d26a186c207be21808ccf2bd7363993922022b110a31f45c86fbc00914b6c5dbd98504c7be7f94e53b158a7aabe322b95695775b82b17f933928bf2cfded7d869698c6cf274b0c2f9c58feff0e8fb2b3ba82e845d8e82149b37a7f3ce7791ec0e165d1e8b853e4224a4fbd1a58127961bbf0c2eddf73fe99674549e61ec05ba1fe8fdaf3e631e6daaddc75a96eccbe14fe2f12a1c056082b8aada2fcf08a4be57f88606e981b11d5a5a56d3926ad6092a5aa804eb342fbeee3d60b1e61e8824e7eed9b8fc8a44188f30ab50de839e20890d0ca86a72ab18a2022e698a2a99d7b9e31bc0c6c7e3384ceea4e43895d923babbfa9ae7a4ba65eb94203927bf1b000f10fbe11621726cb132c1f54ff4892bc4e42659ca7d6a9d97d4a2d1e098b72a8e86058095776ffa2ba811e7886ad403d7bd08bd5ed2fdccab716f0cb685463b6f6f6bb8c467a620863c93b817c4cf28ef431d57350fb95b23fcf05171f5b2d79788ee247e21cb33d50a6f4d67dda8eb05b566e7c14cfd5ee52a5e37a67c4bd9a698cd9e9c2adc10a9c3b89090337f7c3702e2c7d5c01873944853eb833a7aafbc42c48481c7d6d044b1085993f78d04dc8715c28cfe818cab5d01bac5c9258155c257fa025558add2301e77e019d43d4469c977f038daff0ebacffa54001e32f2954a2e019abf440036b1fc9e8a8c3b1ba4e97042dd2d953f38a7e104adbea73a7b08584d912298af5cf8de66a8420b27c87b4ecd434c0271f63703bb91b234d001052c8c55ccaddbcdf9444771fb30b0a9c6fe1d290cb3727399d33816f426ac3b37be50942cef57c679142f3f3f055d3efd52d966ee

// d4a3731ae72140a58b39a2e3356a5971b489402ec60f70e1b6f53241a20c225c500f8960fa79def2505c09422d93a658c193fbb660f86e17d3d26e69f6e21e19086b4eacdd6b9bffe06c83c7cda32e255c145df02b31f04ff754dcbfae54da15d079a69ecda1df646660a8ca773745630afc804f72dcbcd60374e45f44ba288681ce34705c9d111eb5d714225463e41602d3c3e4bd4fd1f78e8458748a5f048b40cde6190a163c1e487467b8281a3f96709241cf0486a6bc0a4c39c6cafef2bd388c0df908f1e86f4699c584aa39c86f94111ec5eb9dea30e7c6e89aa477e9b82efb49f089715653aa146e06a4440249256d24b638190a02541c81c183934f30441a72b190992593ccbbd69785afe23d149cf7c5279bc32b878a9f66c005439e6baefca86bfc5c06a5db7e0f916ca1de9dd6a94c6f5c6cd3c92f1dc29708197c3a7567a5c3c5f9e1f43460098e898734cd21c392687b8faf80da32fe66f3467d9cb28c09dacade1e0515e7587d6ed60101ce6e78cbf9d505b6ff0b63b1b7a1875cfc0df606d10e85de74dbb6804099678c05505f3625db279645ab0a03458750c051654dbb6eba6d30a8d4860fe52366e585069d8eb59e185819415996371d4da2be287b8cd89d7e3f09ca14b6e4601d9e8db0a3af584d9e4c0c59595447225a39b5196534825965d8837380ae9d5457ced63cd1597baf1f7104e345d63900ed6ce8
__constant__ const unsigned char CMP[] = {30, 12, 97, 46, 147, 112, 175, 112, 239, 211, 139, 56, 216, 199, 13, 24, 90, 186, 254, 215, 205, 212, 84, 143, 214, 217, 132, 149, 171, 47, 159, 214, 0, 14, 181, 211, 206, 133, 74, 141, 27, 111, 128, 220, 10, 163, 123, 20, 55, 162, 193, 190, 131, 134, 204, 233, 205, 172, 188, 8, 52, 217, 37, 243, 140, 103, 214, 184, 67, 142, 58, 168, 141, 158, 173, 202, 251, 208, 19, 105, 131, 29, 219, 155, 234, 149, 220, 38, 115, 42, 251, 151, 241, 9, 129, 183, 141, 146, 161, 105, 93, 129, 138, 82, 20, 31, 190, 127, 108, 209};

__global__	void brute() {
	// int control_bit = constant_bits[0];
	// ll state = threadIdx.x + blockIdx.x * (blockIdx.y + gridDim.x * (blockDim.y + gridDim.y * blockDim.z));

	// ll time = 0x000000068d582d1;
	ll time = 1758828330;
	// ll pid = 0x0000000000f9e85 - 4000 + blockIdx.z;
	ll pid = blockIdx.z;
	// ll addr = 0x00007fe27f9216c0;
	//        0x00007fe27f9216c0
	ll addr = 0x00007000000006c0 + ((ll)(threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y))) << 12);
	ll state = ((time << 32) ^ pid) ^ addr;
	// printf("Initial state: %llx\n", state);
	// printf("Trying! Time: %llx\n", (threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y))));
	//         0x00007fe27f9216c0
	// if (addr > 0x000070e0000006c0)
	// 	printf("Trying! Addr: %llx\n", addr);

	ll v10 = STEP(state);
	ll v13 = STEP(v10);
	ll byte = STEP(v13);
	ll v17 = 0x4F6CDD1D * (int)byte % 0x201u + 384;

	if (v17 != 857)
		return ;

	ll idx = 0;
	unsigned char buf[0x201u+384];
	do {
		byte ^= STEP(byte) ^ byte;
		buf[idx++] = 0x1D * byte;
	} while (v17 > (int)idx);
	
	// for (int i = 0; i < v17; i++) {
	// 	printf("%02x", buf[i]);
	// }
	// printf("\n");

	ll i = 0;
	ll *s1 = (ll *)buf, *s2 = (ll *)CMP;
	while (i < 4) {
		if (s1[i] != s2[i])
			return ;
		++i;
	}

	printf("Found! Time: %llx\n", time);
	printf("Found! Pid: %llx\n", pid);
	printf("Found! Addr: %llx\n", addr);
	printf("Found! State: %llx\n", state);
}

int main(void) {
	clock_t			start;
	clock_t			stop;

	// dim3	blocks(256, 256, 256);
	// dim3	threads(256);
	dim3	blocks(256*4*2, 256*4*2, 512*2);
	dim3	threads(256, 4);
	// dim3	blocks(256*4*2, 256*4*2, 1);
	// dim3	threads(256, 4);
	// dim3	blocks(1);
	// dim3	threads(1);

	start = clock();

	brute<<<blocks, threads>>>();
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	stop = clock();
	std::cerr << "Took: " << ((double)(stop - start)) / CLOCKS_PER_SEC << "\n";

	return (0);
}
